#include "hip/hip_runtime.h"
#include "NormalDistributionsTransform.h"
#include "debug.h"
#include <cmath>
#include <iostream>
#include <pcl/common/transforms.h>

#include <tf/tf.h>
#include <tf/transform_broadcaster.h>
#include <tf/transform_datatypes.h>
#include <tf/transform_listener.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>

#include <ros/ros.h>

#include "NormalDistributionsTransform_culib.cuh"
#define V2_ 1
#define FOUR_COUNT 1
std::chrono::time_point<std::chrono::system_clock> debugtime;
double debugtime_ans = 0.0;
namespace gpu {

GNormalDistributionsTransform::GNormalDistributionsTransform()
{
	GRegistration::GRegistration();

	gauss_d1_ = gauss_d2_ = 0;
	outlier_ratio_ = 0.55;
	step_size_ = 0.1;
	resolution_ = 1.0;
//	resolution_ = 1.0f;
	trans_probability_ = 0;

	double gauss_c1, gauss_c2, gauss_d3;

	// Initializes the guassian fitting parameters (eq. 6.8) [Magnusson 2009]
	gauss_c1 = 10.0 * (1 - outlier_ratio_);
	gauss_c2 = outlier_ratio_ / pow (resolution_, 3);
	gauss_d3 = -log (gauss_c2);
	gauss_d1_ = -log ( gauss_c1 + gauss_c2 ) - gauss_d3;
	gauss_d2_ = -2 * log ((-log ( gauss_c1 * exp ( -0.5 ) + gauss_c2 ) - gauss_d3) / gauss_d1_);

	transformation_epsilon_ = 0.1;
	max_iterations_ = 35;

	j_ang_a_ = MatrixHost(3, 1);
	j_ang_b_ = MatrixHost(3, 1);
	j_ang_c_ = MatrixHost(3, 1);
	j_ang_d_ = MatrixHost(3, 1);
	j_ang_e_ = MatrixHost(3, 1);
	j_ang_f_ = MatrixHost(3, 1);
	j_ang_g_ = MatrixHost(3, 1);
	j_ang_h_ = MatrixHost(3, 1);

	h_ang_a2_ = MatrixHost(3, 1);
	h_ang_a3_ = MatrixHost(3, 1);
	h_ang_b2_ = MatrixHost(3, 1);
	h_ang_b3_ = MatrixHost(3, 1);
	h_ang_c2_ = MatrixHost(3, 1);
	h_ang_c3_ = MatrixHost(3, 1);
	h_ang_d1_ = MatrixHost(3, 1);
	h_ang_d2_ = MatrixHost(3, 1);
	h_ang_d3_ = MatrixHost(3, 1);
	h_ang_e1_ = MatrixHost(3, 1);
	h_ang_e2_ = MatrixHost(3, 1);
	h_ang_e3_ = MatrixHost(3, 1);
	h_ang_f1_ = MatrixHost(3, 1);
	h_ang_f2_ = MatrixHost(3, 1);
	h_ang_f3_ = MatrixHost(3, 1);

	dj_ang_a_ = MatrixDevice(3, 1);
	dj_ang_b_ = MatrixDevice(3, 1);
	dj_ang_c_ = MatrixDevice(3, 1);
	dj_ang_d_ = MatrixDevice(3, 1);
	dj_ang_e_ = MatrixDevice(3, 1);
	dj_ang_f_ = MatrixDevice(3, 1);
	dj_ang_g_ = MatrixDevice(3, 1);
	dj_ang_h_ = MatrixDevice(3, 1);

	dh_ang_a2_ = MatrixDevice(3, 1);
	dh_ang_a3_ = MatrixDevice(3, 1);
	dh_ang_b2_ = MatrixDevice(3, 1);
	dh_ang_b3_ = MatrixDevice(3, 1);
	dh_ang_c2_ = MatrixDevice(3, 1);
	dh_ang_c3_ = MatrixDevice(3, 1);
	dh_ang_d1_ = MatrixDevice(3, 1);
	dh_ang_d2_ = MatrixDevice(3, 1);
	dh_ang_d3_ = MatrixDevice(3, 1);
	dh_ang_e1_ = MatrixDevice(3, 1);
	dh_ang_e2_ = MatrixDevice(3, 1);
	dh_ang_e3_ = MatrixDevice(3, 1);
	dh_ang_f1_ = MatrixDevice(3, 1);
	dh_ang_f2_ = MatrixDevice(3, 1);
	dh_ang_f3_ = MatrixDevice(3, 1);

	real_iterations_ = 0;
}

GNormalDistributionsTransform::~GNormalDistributionsTransform()
{
	dj_ang_a_.memFree();
	dj_ang_b_.memFree();
	dj_ang_c_.memFree();
	dj_ang_d_.memFree();
	dj_ang_e_.memFree();
	dj_ang_f_.memFree();
	dj_ang_g_.memFree();
	dj_ang_h_.memFree();

	dh_ang_a2_.memFree();
	dh_ang_a3_.memFree();
	dh_ang_b2_.memFree();
	dh_ang_b3_.memFree();
	dh_ang_c2_.memFree();
	dh_ang_c3_.memFree();
	dh_ang_d1_.memFree();
	dh_ang_d2_.memFree();
	dh_ang_d3_.memFree();
	dh_ang_e1_.memFree();
	dh_ang_e2_.memFree();
	dh_ang_e3_.memFree();
	dh_ang_f1_.memFree();
	dh_ang_f2_.memFree();
	dh_ang_f3_.memFree();

	voxel_grid_.~GVoxelGrid();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////
	void GNormalDistributionsTransform::setStepSize(double step_size)
	{
		step_size_ = step_size;
	}

	void GNormalDistributionsTransform::setResolution(float resolution)
	{
//fprintf(stderr, "setresolution to %f **********************************************************\n", resolution);
		resolution_ = resolution;
//fprintf(stderr, "resolotion_ = %f **********************************************************\n", resolution_);
	}

	void GNormalDistributionsTransform::setOutlierRatio(double olr)
	{
		outlier_ratio_ = olr;
	}

	double GNormalDistributionsTransform::getStepSize()
	{
		return step_size_;
	}

	float GNormalDistributionsTransform::getResolution()
	{
		return resolution_;
	}

	double GNormalDistributionsTransform::getOutlierRatio()
	{
		return outlier_ratio_;
	}

	double GNormalDistributionsTransform::getTransformationProbability()
	{
		return trans_probability_;
	}

	int GNormalDistributionsTransform::getRealIterations() { return real_iterations_; }
///////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void gpuSum(T *input, int size, int half_size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = idx; i < half_size; i += stride) {
		input[i] += (half_size < size) ? input[i + half_size] : 0;
	}
}

void GNormalDistributionsTransform::setInputTarget(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	// Copy input map data from the host memory to the GPU memory
	GRegistration::setInputTarget(input);

	// Build the voxel grid
	if (target_points_number_ != 0) {
		voxel_grid_.setLeafSize(resolution_, resolution_, resolution_);
		voxel_grid_.setInput(target_x_, target_y_, target_z_, target_points_number_);
	}
}

void GNormalDistributionsTransform::setInputTarget(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	// Copy input map data from the host memory to the GPU memory
	GRegistration::setInputTarget(input);

	// Build the voxel grid
	if (target_points_number_ != 0) {
		voxel_grid_.setLeafSize(1.0, 1.0, 1.0);
		voxel_grid_.setInput(target_x_, target_y_, target_z_, target_points_number_);
	}
}

void GNormalDistributionsTransform::computeTransformation(Eigen::Matrix<float, 4, 4> &guess)
{
	struct timeval start, end;

	nr_iterations_ = 0;
	converged_ = false;

	double gauss_c1, gauss_c2, gauss_d3;

	gauss_c1 = 10 * ( 1 - outlier_ratio_);
	gauss_c2 = outlier_ratio_ / pow(resolution_, 3);
	gauss_d3 = - log(gauss_c2);
	gauss_d1_ = -log(gauss_c1 + gauss_c2) - gauss_d3;
	gauss_d2_ = -2 * log((-log(gauss_c1 * exp(-0.5) + gauss_c2) - gauss_d3) / gauss_d1_);
	if (guess != Eigen::Matrix4f::Identity()) {
		final_transformation_ = guess;
		transformPointCloud(x_, y_, z_, trans_x_, trans_y_, trans_z_, points_number_, guess);
	}
	Eigen::Transform<float, 3, Eigen::Affine, Eigen::ColMajor> eig_transformation;
	eig_transformation.matrix() = final_transformation_;

	Eigen::Matrix<double, 6, 1> p, delta_p, score_gradient;
	Eigen::Vector3f init_translation = eig_transformation.translation();

	tfScalar roll_, pitch_, yaw_;
    
	tf::Matrix3x3 mat_l;  // localizer
        mat_l.setValue(static_cast<double>(final_transformation_(0, 0)), static_cast<double>(final_transformation_(0, 1)), static_cast<double>(final_transformation_(0, 2)),
                   static_cast<double>(final_transformation_(1, 0)), static_cast<double>(final_transformation_(1, 1)), static_cast<double>(final_transformation_(1, 2)),
                   static_cast<double>(final_transformation_(2, 0)), static_cast<double>(final_transformation_(2, 1)), static_cast<double>(final_transformation_(2, 2)));
	mat_l.getRPY(roll_, pitch_, yaw_, 1);

	p << init_translation(0), init_translation(1), init_translation(2), roll_, pitch_, yaw_;
	Eigen::Matrix<double, 6, 6> hessian;

	double score = 0;
	double delta_p_norm;

	gettimeofday(&start, NULL);
	score = computeDerivatives(score_gradient, hessian, trans_x_, trans_y_, trans_z_, points_number_, p);
	gettimeofday(&end, NULL);
	gettimeofday(&start, NULL);
	while (!converged_) {
		previous_transformation_ = transformation_;
		
		Eigen::JacobiSVD<Eigen::Matrix<double, 6, 6>> sv(hessian, Eigen::ComputeFullU | Eigen::ComputeFullV);

		delta_p = sv.solve(-score_gradient);

		delta_p_norm = delta_p.norm();

		if (delta_p_norm == 0 || delta_p_norm != delta_p_norm) {

			trans_probability_ = score / static_cast<double>(points_number_);
			converged_ = delta_p_norm == delta_p_norm;
			return;
		}

		delta_p.normalize();

		delta_p_norm = computeStepLengthMT(p, delta_p, delta_p_norm, step_size_, 
			transformation_epsilon_ / 2, score, score_gradient, hessian, 
			trans_x_, trans_y_, trans_z_, points_number_);
		delta_p *= delta_p_norm;

#ifdef FOUR_COUNT
Eigen::Matrix3f R;  
    R = Eigen::AngleAxis<float>(static_cast<float>(delta_p(3)), Eigen::Vector3f::UnitX())  
        * Eigen::AngleAxis<float>(static_cast<float>(delta_p(4)), Eigen::Vector3f::UnitY())  
        * Eigen::AngleAxis<float>(static_cast<float>(delta_p(5)), Eigen::Vector3f::UnitZ());  
    Eigen::Quaternionf q;  
    q = R;

    double a[3][3];
	a[0][0] = 1-2*(q.y()*q.y()+q.z()*q.z());
	a[0][1] = 2*(q.x()*q.y()-q.z()*q.w());
	a[0][2] = 2*(q.x()*q.z()+q.y()*q.w());
	a[1][0] = 2*(q.x()*q.y()+q.z()*q.w());
	a[1][1] = 1-2*(q.x()*q.x()+q.z()*q.z());
	a[1][2] = 2*(q.y()*q.z()-q.x()*q.w());
	a[2][0] = 2*(q.x()*q.z()-q.y()*q.w());
	a[2][1] = 2*(q.y()*q.z()+q.x()*q.w());
	a[2][2] = 1-2*(q.x()*q.x()+q.y()*q.y());

transformation_ << a[0][0],a[0][1],a[0][2],static_cast<float>(delta_p(0)),a[1][0],a[1][1],a[1][2],static_cast<float>(delta_p(1)),
					a[2][0],a[2][1],a[2][2],static_cast<float>(delta_p(2)),0,0,0,1;
#endif

		p = p + delta_p;

		//Not update visualizer
		nr_iterations_++;
		if (nr_iterations_ > 15 || (nr_iterations_ && (std::fabs(delta_p_norm) < transformation_epsilon_)))//max_iterations_
			converged_ = true;
	}
	gettimeofday(&end, NULL);

	char buffer_1026[20];
	sprintf(buffer_1026,"%d:%d;\n",points_number_,nr_iterations_);
	save_debug_data(buffer_1026,0,1);

	trans_probability_ = score / static_cast<double>(points_number_);
}


double GNormalDistributionsTransform::computeDerivatives(Eigen::Matrix<double, 6, 1> &score_gradient, Eigen::Matrix<double, 6, 6> &hessian,
														float *trans_x, float *trans_y, float *trans_z,
														int points_num, Eigen::Matrix<double, 6, 1> pose, bool compute_hessian)
{
	MatrixHost p(6, 1);

	for (int i = 0; i < 6; i++) {
		p(i) = pose(i, 0);
	}

	score_gradient.setZero ();
	hessian.setZero ();

	//Compute Angle Derivatives
	computeAngleDerivatives(p);

	//Radius Search
	int *valid_points, *voxel_id, *starting_voxel_id;
	int valid_voxel_num, valid_points_num;

	valid_points = voxel_id = starting_voxel_id = NULL;


	voxel_grid_.radiusSearch(trans_x, trans_y, trans_z, points_num, resolution_, INT_MAX, &valid_points, &starting_voxel_id, &voxel_id, &valid_voxel_num, &valid_points_num);

	double *covariance = voxel_grid_.getCovarianceList();
	double *inverse_covariance = voxel_grid_.getInverseCovarianceList();
	double *centroid = voxel_grid_.getCentroidList();
	int *points_per_voxel = voxel_grid_.getPointsPerVoxelList();
	int voxel_num = voxel_grid_.getVoxelNum();

	if (valid_points_num == 0)
		return 0;

	//Update score gradient and hessian matrix

	double *gradients, *hessians, *point_gradients, *point_hessians, *score;

	checkCudaErrors(hipMalloc(&gradients, sizeof(double) * valid_points_num * 6));
	checkCudaErrors(hipMemset(gradients, 0, sizeof(double) * valid_points_num * 6));

	checkCudaErrors(hipMalloc(&hessians, sizeof(double) * valid_points_num * 6 * 6));
	checkCudaErrors(hipMemset(hessians, 0, sizeof(double) * valid_points_num * 6 * 6));

	checkCudaErrors(hipMalloc(&point_gradients, sizeof(double) * valid_points_num * 3 * 6));
	checkCudaErrors(hipMemset(point_gradients, 0, sizeof(double) * valid_points_num * 3 * 6));

	checkCudaErrors(hipMalloc(&point_hessians, sizeof(double) * valid_points_num * 18 * 6));
	checkCudaErrors(hipMemset(point_hessians, 0, sizeof(double) * valid_points_num * 18 * 6));

	checkCudaErrors(hipMalloc(&score, sizeof(double) * valid_points_num));

	int block_x;
#ifdef USING_HP
	block_x = (valid_points_num > (BLOCK_SIZE_X)) ? (BLOCK_SIZE_X) : valid_points_num;//gpu_up
#else
	block_x = (valid_points_num > (BLOCK_SIZE_X/2)) ? (BLOCK_SIZE_X/2) : valid_points_num;
#endif
	int grid_x = (valid_points_num - 1) / block_x + 1;

	dim3 grid;
	computePointGradients0<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dj_ang_a_, dj_ang_b_, dj_ang_c_, dj_ang_d_,
												point_gradients);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	computePointGradients1<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dj_ang_e_, dj_ang_f_, dj_ang_g_, dj_ang_h_,
												point_gradients);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	if (compute_hessian) {
		computePointHessian0<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_a2_, dh_ang_a3_,
												dh_ang_b2_, dh_ang_b3_,
												point_hessians);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computePointHessian1<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_c2_, dh_ang_c3_,
												dh_ang_d1_, dh_ang_d2_, dh_ang_d3_,
												point_hessians);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computePointHessian2<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_e1_, dh_ang_e2_, dh_ang_e3_,
												dh_ang_f1_, dh_ang_f2_, dh_ang_f3_,
												point_hessians);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui
	}
	checkCudaErrors(hipDeviceSynchronize());

	double *tmp_hessian;

	checkCudaErrors(hipMalloc(&tmp_hessian, sizeof(double) * valid_voxel_num * 6));

	double *e_x_cov_x;

	checkCudaErrors(hipMalloc(&e_x_cov_x, sizeof(double) * valid_voxel_num));

	double *cov_dxd_pi;

	checkCudaErrors(hipMalloc(&cov_dxd_pi, sizeof(double) * valid_voxel_num * 3 * 6));

	computeExCovX<<<grid_x, block_x>>>(trans_x, trans_y, trans_z, valid_points,
										starting_voxel_id, voxel_id, valid_points_num,
										centroid, inverse_covariance, voxel_num,
										gauss_d1_, gauss_d2_,
										e_x_cov_x);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	computeScoreList<<<grid_x, block_x>>>(starting_voxel_id, voxel_id, valid_points_num, e_x_cov_x, gauss_d1_, score);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	int block_x2 = (valid_voxel_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : valid_voxel_num;
	int grid_x2 = (valid_voxel_num - 1) / block_x2 + 1;

	updateExCovX<<<grid_x2, block_x2>>>(e_x_cov_x, gauss_d2_, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	grid.x = grid_x;
	grid.y = 3;
	grid.z = 6;

	computeCovDxdPi<<<grid, block_x>>>(valid_points, starting_voxel_id, voxel_id, valid_points_num,
											inverse_covariance, voxel_num,
											gauss_d1_, gauss_d2_, point_gradients,
											cov_dxd_pi, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	grid.x = grid_x;
	grid.y = 6;
	grid.z = 1;
	computeScoreGradientList<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, voxel_num, e_x_cov_x,
													cov_dxd_pi, gauss_d1_, valid_voxel_num, gradients);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());//added by panrui

	if (compute_hessian) {
		grid.y = 6;
		grid.z = 1;
		computeHessianListS0<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_gradients, tmp_hessian, valid_voxel_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS1<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_gradients, tmp_hessian, valid_voxel_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS2<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_gradients, tmp_hessian, valid_voxel_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		grid.z = 6;
		computeHessianListS3<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, voxel_num,
													gauss_d1_, gauss_d2_,
													hessians,
													e_x_cov_x, tmp_hessian, cov_dxd_pi,
													valid_voxel_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS4<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_hessians, hessians,
													e_x_cov_x);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS5<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
														starting_voxel_id, voxel_id, valid_points_num,
														centroid, inverse_covariance, voxel_num,
														gauss_d1_, gauss_d2_,
														point_hessians, hessians,
														e_x_cov_x);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS6<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
														starting_voxel_id, voxel_id, valid_points_num,
														centroid, inverse_covariance, voxel_num,
														gauss_d1_, gauss_d2_,
														point_hessians, hessians,
														e_x_cov_x);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		computeHessianListS7<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													gauss_d1_, gauss_d2_,
													point_gradients, hessians,
													e_x_cov_x, cov_dxd_pi, valid_voxel_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui
	}
	int full_size = valid_points_num;
	int half_size = (full_size - 1) / 2 + 1;
	while (full_size > 1) {
		block_x = (half_size > BLOCK_SIZE_X) ? BLOCK_SIZE_X : half_size;
		grid_x = (half_size - 1) / block_x + 1;

		grid.x = grid_x;
		grid.y = 1;
		grid.z = 6;
		matrixSum<<<grid, block_x>>>(gradients, full_size, half_size, 1, 6, valid_points_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		grid.y = 6;
		matrixSum<<<grid, block_x>>>(hessians, full_size, half_size, 6, 6, valid_points_num);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		sumScore<<<grid_x, block_x>>>(score, full_size, half_size);
		checkCudaErrors(hipGetLastError());
//		checkCudaErrors(hipDeviceSynchronize());//added by panrui

		full_size = half_size;
		half_size = (full_size - 1) / 2 + 1;
	}

	checkCudaErrors(hipDeviceSynchronize());

	MatrixDevice dgrad(1, 6, valid_points_num, gradients), dhess(6, 6, valid_points_num, hessians);
	MatrixHost hgrad(1, 6), hhess(6, 6);

	hgrad.moveToHost(dgrad);
	hhess.moveToHost(dhess);
	for (int i = 0; i < 6; i++) {
		score_gradient(i) = hgrad(i);
	}
	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 6; j++) {
			hessian(i, j) = hhess(i, j);
		}
	}

	double score_inc;

	checkCudaErrors(hipMemcpy(&score_inc, score, sizeof(double), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(gradients));
	checkCudaErrors(hipFree(hessians));
	checkCudaErrors(hipFree(point_hessians));
	checkCudaErrors(hipFree(point_gradients));
	checkCudaErrors(hipFree(score));

	checkCudaErrors(hipFree(tmp_hessian));

	checkCudaErrors(hipFree(e_x_cov_x));
	checkCudaErrors(hipFree(cov_dxd_pi));

	if (valid_points != NULL)
		checkCudaErrors(hipFree(valid_points));

	if (voxel_id != NULL)
		checkCudaErrors(hipFree(voxel_id));

	if (starting_voxel_id != NULL)
		checkCudaErrors(hipFree(starting_voxel_id));

	return score_inc;
}

void GNormalDistributionsTransform::computeAngleDerivatives(MatrixHost pose, bool compute_hessian)
{
	double cx, cy, cz, sx, sy, sz;

	if (fabs(pose(3)) < 10e-5) {
		cx = 1.0;
		sx = 0.0;
	} else {
		cx = cos(pose(3));
		sx = sin(pose(3));
	}

	if (fabs(pose(4)) < 10e-5) {
		cy = 1.0;
		sy = 0.0;
	} else {
		cy = cos(pose(4));
		sy = sin(pose(4));
	}

	if (fabs(pose(5)) < 10e-5) {
		cz = 1.0;
		sz = 0.0;
	} else {
		cz = cos(pose(5));
		sz = sin(pose(5));
	}

	j_ang_a_(0) = -sx * sz + cx * sy * cz;
	j_ang_a_(1) = -sx * cz - cx * sy * sz;
	j_ang_a_(2) = -cx * cy;

	j_ang_b_(0) = cx * sz + sx * sy * cz;
	j_ang_b_(1) = cx * cz - sx * sy * sz;
	j_ang_b_(2) = -sx * cy;

	j_ang_c_(0) = -sy * cz;
	j_ang_c_(1) = sy * sz;
	j_ang_c_(2) = cy;

	j_ang_d_(0) = sx * cy * cz;
	j_ang_d_(1) = -sx * cy * sz;
	j_ang_d_(2) = sx * sy;

	j_ang_e_(0) = -cx * cy * cz;
	j_ang_e_(1) = cx * cy * sz;
	j_ang_e_(2) = -cx * sy;

	j_ang_f_(0) = -cy * sz;
	j_ang_f_(1) = -cy * cz;
	j_ang_f_(2) = 0;

	j_ang_g_(0) = cx * cz - sx * sy * sz;
	j_ang_g_(1) = -cx * sz - sx * sy * cz;
	j_ang_g_(2) = 0;

	j_ang_h_(0) = sx * cz + cx * sy * sz;
	j_ang_h_(1) = cx * sy * cz - sx * sz;
	j_ang_h_(2) = 0;

	j_ang_a_.moveToGpu(dj_ang_a_);
	j_ang_b_.moveToGpu(dj_ang_b_);
	j_ang_c_.moveToGpu(dj_ang_c_);
	j_ang_d_.moveToGpu(dj_ang_d_);
	j_ang_e_.moveToGpu(dj_ang_e_);
	j_ang_f_.moveToGpu(dj_ang_f_);
	j_ang_g_.moveToGpu(dj_ang_g_);
	j_ang_h_.moveToGpu(dj_ang_h_);

	if (compute_hessian) {
		h_ang_a2_(0) = -cx * sz - sx * sy * cz;
		h_ang_a2_(1) = -cx * cz + sx * sy * sz;
		h_ang_a2_(2) = sx * cy;

		h_ang_a3_(0) = -sx * sz + cx * sy * cz;
		h_ang_a3_(1) = -cx * sy * sz - sx * cz;
		h_ang_a3_(2) = -cx * cy;

		h_ang_b2_(0) = cx * cy * cz;
		h_ang_b2_(1) = -cx * cy * sz;
		h_ang_b2_(2) = cx * sy;

		h_ang_b3_(0) = sx * cy * cz;
		h_ang_b3_(1) = -sx * cy * sz;
		h_ang_b3_(2) = sx * sy;

		h_ang_c2_(0) = -sx * cz - cx * sy * sz;
		h_ang_c2_(1) = sx * sz - cx * sy * cz;
		h_ang_c2_(2) = 0;

		h_ang_c3_(0) = cx * cz - sx * sy * sz;
		h_ang_c3_(1) = -sx * sy * cz - cx * sz;
		h_ang_c3_(2) = 0;

		h_ang_d1_(0) = -cy * cz;
		h_ang_d1_(1) = cy * sz;
		h_ang_d1_(2) = sy;

		h_ang_d2_(0) = -sx * sy * cz;
		h_ang_d2_(1) = sx * sy * sz;
		h_ang_d2_(2) = sx * cy;

		h_ang_d3_(0) = cx * sy * cz;
		h_ang_d3_(1) = -cx * sy * sz;
		h_ang_d3_(2) = -cx * cy;

		h_ang_e1_(0) = sy * sz;
		h_ang_e1_(1) = sy * cz;
		h_ang_e1_(3) = 0;

		h_ang_e2_(0) = -sx * cy * sz;
		h_ang_e2_(1) = -sx * cy * cz;
		h_ang_e2_(2) = 0;

		h_ang_e3_(0) = cx * cy * sz;
		h_ang_e3_(1) = cx * cy * cz;
		h_ang_e3_(2) = 0;

		h_ang_f1_(0) = -cy * cz;
		h_ang_f1_(1) = cy * sz;
		h_ang_f1_(2) = 0;

		h_ang_f2_(0) = -cx * sz - sx * sy * cz;
		h_ang_f2_(1) = -cx * cz + sx * sy * sz;
		h_ang_f2_(2) = 0;

		h_ang_f3_(0) = -sx * sz + cx * sy * cz;
		h_ang_f3_(1) = -cx * sy * sz - sx * cz;
		h_ang_f3_(2) = 0;

		h_ang_a2_.moveToGpu(dh_ang_a2_);
		h_ang_a3_.moveToGpu(dh_ang_a3_);
		h_ang_b2_.moveToGpu(dh_ang_b2_);
		h_ang_b3_.moveToGpu(dh_ang_b3_);
		h_ang_c2_.moveToGpu(dh_ang_c2_);
		h_ang_c3_.moveToGpu(dh_ang_c3_);
		h_ang_d1_.moveToGpu(dh_ang_d1_);
		h_ang_d2_.moveToGpu(dh_ang_d2_);
		h_ang_d3_.moveToGpu(dh_ang_d3_);
		h_ang_e1_.moveToGpu(dh_ang_e1_);
		h_ang_e2_.moveToGpu(dh_ang_e2_);
		h_ang_e3_.moveToGpu(dh_ang_e3_);
		h_ang_f1_.moveToGpu(dh_ang_f1_);
		h_ang_f2_.moveToGpu(dh_ang_f2_);
		h_ang_f3_.moveToGpu(dh_ang_f3_);
	}

}

void GNormalDistributionsTransform::transformPointCloud(float *in_x, float *in_y, float *in_z,
														float *trans_x, float *trans_y, float *trans_z,
														int points_number, Eigen::Matrix<float, 4, 4> transform)
{
	Eigen::Transform<float, 3, Eigen::Affine> t(transform);

	MatrixHost htrans(3, 4);
	MatrixDevice dtrans(3, 4);

	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 4; j++) {
			htrans(i, j) = t(i, j);
		}
	}

	htrans.moveToGpu(dtrans);

	if (points_number > 0) {

//added by panrui
		int block_x;
#ifdef USING_HP
		block_x = (points_number <= (BLOCK_SIZE_X)) ? points_number : (BLOCK_SIZE_X);//gpu_up
#else
		block_x = (points_number <= (BLOCK_SIZE_X/2)) ? points_number : (BLOCK_SIZE_X/2);
#endif
		int grid_x = (points_number - 1) / block_x + 1;

		gpuTransform<<<grid_x, block_x >>>(in_x, in_y, in_z, trans_x, trans_y, trans_z, points_number, dtrans);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}

	dtrans.memFree();
}

double GNormalDistributionsTransform::computeStepLengthMT(const Eigen::Matrix<double, 6, 1> &x, Eigen::Matrix<double, 6, 1> &step_dir,
															double step_init, double step_max, double step_min, double &score,
															Eigen::Matrix<double, 6, 1> &score_gradient, Eigen::Matrix<double, 6, 6> &hessian,
															float *trans_x, float *trans_y, float *trans_z, int points_num)
{
	double phi_0 = -score;
	double d_phi_0 = -(score_gradient.dot(step_dir));

	Eigen::Matrix<double, 6, 1> x_t;

	if (d_phi_0 >= 0) {
		if (d_phi_0 == 0)
			return 0;
		else {
			d_phi_0 *= -1;
			step_dir *= -1;
		}
	}

	int max_step_iterations = 10;
	int step_iterations = 0;


	double mu = 1.e-4;
	double nu = 0.9;
	double a_l = 0, a_u = 0;

	double f_l = auxilaryFunction_PsiMT(a_l, phi_0, phi_0, d_phi_0, mu);
	double g_l = auxilaryFunction_dPsiMT(d_phi_0, d_phi_0, mu);

	double f_u = auxilaryFunction_PsiMT(a_u, phi_0, phi_0, d_phi_0, mu);
	double g_u = auxilaryFunction_dPsiMT(d_phi_0, d_phi_0, mu);

	bool interval_converged = (step_max - step_min) > 0, open_interval = true;

	double a_t = step_init;
	a_t = std::min(a_t, step_max);
	a_t = std::max(a_t, step_min);

	x_t = x + step_dir * a_t;

#ifdef FOUR_COUNT
Eigen::Matrix3f R2;  
    R2 = Eigen::AngleAxis<float>(static_cast<float>(x_t(3)), Eigen::Vector3f::UnitX())  
        * Eigen::AngleAxis<float>(static_cast<float>(x_t(4)), Eigen::Vector3f::UnitY())  
        * Eigen::AngleAxis<float>(static_cast<float>(x_t(5)), Eigen::Vector3f::UnitZ());  
    Eigen::Quaternionf q2;  
    q2 = R2;

    double a[3][3];
	a[0][0] = 1-2*(q2.y()*q2.y()+q2.z()*q2.z());
	a[0][1] = 2*(q2.x()*q2.y()-q2.z()*q2.w());
	a[0][2] = 2*(q2.x()*q2.z()+q2.y()*q2.w());
	a[1][0] = 2*(q2.x()*q2.y()+q2.z()*q2.w());
	a[1][1] = 1-2*(q2.x()*q2.x()+q2.z()*q2.z());
	a[1][2] = 2*(q2.y()*q2.z()-q2.x()*q2.w());
	a[2][0] = 2*(q2.x()*q2.z()-q2.y()*q2.w());
	a[2][1] = 2*(q2.y()*q2.z()+q2.x()*q2.w());
	a[2][2] = 1-2*(q2.x()*q2.x()+q2.y()*q2.y());

final_transformation_ << a[0][0],a[0][1],a[0][2],static_cast<float>(x_t(0)),a[1][0],a[1][1],a[1][2],static_cast<float>(x_t(1)),
					a[2][0],a[2][1],a[2][2],static_cast<float>(x_t(2)),0,0,0,1;
#endif

	transformPointCloud(x_, y_, z_, trans_x, trans_y, trans_z, points_num, final_transformation_);

	score = computeDerivatives(score_gradient, hessian, trans_x, trans_y, trans_z, points_num, x_t);

	double phi_t = -score;
	double d_phi_t = -(score_gradient.dot(step_dir));
	double psi_t = auxilaryFunction_PsiMT(a_t, phi_t, phi_0, d_phi_0, mu);
	double d_psi_t = auxilaryFunction_dPsiMT(d_phi_t, d_phi_0, mu);

	while (!interval_converged && step_iterations < max_step_iterations && !(psi_t <= 0 && d_phi_t <= -nu * d_phi_0)) {
		if (open_interval) {
			a_t = trialValueSelectionMT(a_l, f_l, g_l, a_u, f_u, g_u, a_t, psi_t, d_psi_t);
		} else {
			a_t = trialValueSelectionMT(a_l, f_l, g_l, a_u, f_u, g_u, a_t, phi_t, d_phi_t);
		}

		a_t = (a_t < step_max) ? a_t : step_max;
		a_t = (a_t > step_min) ? a_t : step_min;
		
		x_t = x + step_dir * a_t;

#ifdef FOUR_COUNT
	Eigen::Matrix3f R;  
	    R = Eigen::AngleAxis<float>(static_cast<float>(x_t(3)), Eigen::Vector3f::UnitX())  
		* Eigen::AngleAxis<float>(static_cast<float>(x_t(4)), Eigen::Vector3f::UnitY())  
		* Eigen::AngleAxis<float>(static_cast<float>(x_t(5)), Eigen::Vector3f::UnitZ());  
	    Eigen::Quaternionf q;  
	    q = R;

	    double a[3][3];
		a[0][0] = 1-2*(q.y()*q.y()+q.z()*q.z());
		a[0][1] = 2*(q.x()*q.y()-q.z()*q.w());
		a[0][2] = 2*(q.x()*q.z()+q.y()*q.w());
		a[1][0] = 2*(q.x()*q.y()+q.z()*q.w());
		a[1][1] = 1-2*(q.x()*q.x()+q.z()*q.z());
		a[1][2] = 2*(q.y()*q.z()-q.x()*q.w());
		a[2][0] = 2*(q.x()*q.z()-q.y()*q.w());
		a[2][1] = 2*(q.y()*q.z()+q.x()*q.w());
		a[2][2] = 1-2*(q.x()*q.x()+q.y()*q.y());

	final_transformation_ << a[0][0],a[0][1],a[0][2],static_cast<float>(x_t(0)),a[1][0],a[1][1],a[1][2],static_cast<float>(x_t(1)),
						a[2][0],a[2][1],a[2][2],static_cast<float>(x_t(2)),0,0,0,1;
#endif
		transformPointCloud(x_, y_, z_, trans_x, trans_y, trans_z, points_num, final_transformation_);

		score = computeDerivatives(score_gradient, hessian, trans_x, trans_y, trans_z, points_num, x_t, false);

		phi_t -= score;
		d_phi_t -= (score_gradient.dot(step_dir));
		psi_t = auxilaryFunction_PsiMT(a_t, phi_t, phi_0, d_phi_0, mu);
		d_psi_t = auxilaryFunction_dPsiMT(d_phi_t, d_phi_0, mu);

		if (open_interval && (psi_t <= 0 && d_psi_t >= 0)) {
			open_interval = false;

			f_l += phi_0 - mu * d_phi_0 * a_l;
			g_l += mu * d_phi_0;

			f_u += phi_0 - mu * d_phi_0 * a_u;
			g_u += mu * d_phi_0;
		}

		if (open_interval) {
			interval_converged = updateIntervalMT(a_l, f_l, g_l, a_u, f_u, g_u, a_t, psi_t, d_psi_t);
		} else {
			interval_converged = updateIntervalMT(a_l, f_l, g_l, a_u, f_u, g_u, a_t, phi_t, d_phi_t);
		}
		step_iterations++;
	}

	if (step_iterations) {
		computeHessian(hessian, trans_x, trans_y, trans_z, points_num, x_t);
	}

	real_iterations_ += step_iterations;

	return a_t;
}


//Copied from ndt.hpp
double GNormalDistributionsTransform::trialValueSelectionMT (double a_l, double f_l, double g_l,
															double a_u, double f_u, double g_u,
															double a_t, double f_t, double g_t)
{
	// Case 1 in Trial Value Selection [More, Thuente 1994]
	if (f_t > f_l) {
		// Calculate the minimizer of the cubic that interpolates f_l, f_t, g_l and g_t
		// Equation 2.4.52 [Sun, Yuan 2006]
		double z = 3 * (f_t - f_l) / (a_t - a_l) - g_t - g_l;
		double w = std::sqrt (z * z - g_t * g_l);
		// Equation 2.4.56 [Sun, Yuan 2006]
		double a_c = a_l + (a_t - a_l) * (w - g_l - z) / (g_t - g_l + 2 * w);

		// Calculate the minimizer of the quadratic that interpolates f_l, f_t and g_l
		// Equation 2.4.2 [Sun, Yuan 2006]
		double a_q = a_l - 0.5 * (a_l - a_t) * g_l / (g_l - (f_l - f_t) / (a_l - a_t));

		if (std::fabs (a_c - a_l) < std::fabs (a_q - a_l))
		  return (a_c);
		else
		  return (0.5 * (a_q + a_c));
	}
	// Case 2 in Trial Value Selection [More, Thuente 1994]
	else if (g_t * g_l < 0) {
		// Calculate the minimizer of the cubic that interpolates f_l, f_t, g_l and g_t
		// Equation 2.4.52 [Sun, Yuan 2006]
		double z = 3 * (f_t - f_l) / (a_t - a_l) - g_t - g_l;
		double w = std::sqrt (z * z - g_t * g_l);
		// Equation 2.4.56 [Sun, Yuan 2006]
		double a_c = a_l + (a_t - a_l) * (w - g_l - z) / (g_t - g_l + 2 * w);

		// Calculate the minimizer of the quadratic that interpolates f_l, g_l and g_t
		// Equation 2.4.5 [Sun, Yuan 2006]
		double a_s = a_l - (a_l - a_t) / (g_l - g_t) * g_l;

		if (std::fabs (a_c - a_t) >= std::fabs (a_s - a_t))
		  return (a_c);
		else
		  return (a_s);
	}
	// Case 3 in Trial Value Selection [More, Thuente 1994]
	else if (std::fabs (g_t) <= std::fabs (g_l)) {
		// Calculate the minimizer of the cubic that interpolates f_l, f_t, g_l and g_t
		// Equation 2.4.52 [Sun, Yuan 2006]
		double z = 3 * (f_t - f_l) / (a_t - a_l) - g_t - g_l;
		double w = std::sqrt (z * z - g_t * g_l);
		double a_c = a_l + (a_t - a_l) * (w - g_l - z) / (g_t - g_l + 2 * w);

		// Calculate the minimizer of the quadratic that interpolates g_l and g_t
		// Equation 2.4.5 [Sun, Yuan 2006]
		double a_s = a_l - (a_l - a_t) / (g_l - g_t) * g_l;

		double a_t_next;

		if (std::fabs (a_c - a_t) < std::fabs (a_s - a_t))
		  a_t_next = a_c;
		else
		  a_t_next = a_s;

		if (a_t > a_l)
		  return (std::min (a_t + 0.66 * (a_u - a_t), a_t_next));
		else
		  return (std::max (a_t + 0.66 * (a_u - a_t), a_t_next));
	}
	// Case 4 in Trial Value Selection [More, Thuente 1994]
	else {
		// Calculate the minimizer of the cubic that interpolates f_u, f_t, g_u and g_t
		// Equation 2.4.52 [Sun, Yuan 2006]
		double z = 3 * (f_t - f_u) / (a_t - a_u) - g_t - g_u;
		double w = std::sqrt (z * z - g_t * g_u);
		// Equation 2.4.56 [Sun, Yuan 2006]
		return (a_u + (a_t - a_u) * (w - g_u - z) / (g_t - g_u + 2 * w));
	}
}

//Copied from ndt.hpp
double GNormalDistributionsTransform::updateIntervalMT (double &a_l, double &f_l, double &g_l,
														double &a_u, double &f_u, double &g_u,
														double a_t, double f_t, double g_t)
{
  // Case U1 in Update Algorithm and Case a in Modified Update Algorithm [More, Thuente 1994]
	if (f_t > f_l) {
		a_u = a_t;
		f_u = f_t;
		g_u = g_t;
		return (false);
	}
	// Case U2 in Update Algorithm and Case b in Modified Update Algorithm [More, Thuente 1994]
	else if (g_t * (a_l - a_t) > 0) {
		a_l = a_t;
		f_l = f_t;
		g_l = g_t;
		return (false);
	}
	// Case U3 in Update Algorithm and Case c in Modified Update Algorithm [More, Thuente 1994]
	else if (g_t * (a_l - a_t) < 0) {
		a_u = a_l;
		f_u = f_l;
		g_u = g_l;

		a_l = a_t;
		f_l = f_t;
		g_l = g_t;
		return (false);
	}
	// Interval Converged
	else
		return (true);
}

void GNormalDistributionsTransform::computeHessian(Eigen::Matrix<double, 6, 6> &hessian, float *trans_x, float *trans_y, float *trans_z, int points_num, Eigen::Matrix<double, 6, 1> &p)
{
	int *valid_points, *voxel_id, *starting_voxel_id;
	int valid_voxel_num, valid_points_num;
	//Radius Search

	voxel_grid_.radiusSearch(trans_x, trans_y, trans_z, points_num, resolution_, INT_MAX, &valid_points, &starting_voxel_id, &voxel_id, &valid_voxel_num, &valid_points_num);

	double *centroid = voxel_grid_.getCentroidList();
	double *covariance = voxel_grid_.getCovarianceList();
	double *inverse_covariance = voxel_grid_.getInverseCovarianceList();
	int *points_per_voxel = voxel_grid_.getPointsPerVoxelList();
	int voxel_num = voxel_grid_.getVoxelNum();

	if (valid_points_num <= 0)
		return;

	//Update score gradient and hessian matrix
	double *hessians, *point_gradients, *point_hessians;

	checkCudaErrors(hipMalloc(&hessians, sizeof(double) * valid_points_num * 6 * 6));
	checkCudaErrors(hipMemset(hessians, 0, sizeof(double) * valid_points_num * 6 * 6));

	checkCudaErrors(hipMalloc(&point_gradients, sizeof(double) * valid_points_num * 3 * 6));
	checkCudaErrors(hipMemset(point_gradients, 0, sizeof(double) * valid_points_num * 3 * 6));

	checkCudaErrors(hipMalloc(&point_hessians, sizeof(double) * valid_points_num * 18 * 6));
	checkCudaErrors(hipMemset(point_hessians, 0, sizeof(double) * valid_points_num * 18 * 6));

	int block_x = (valid_points_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : valid_points_num;
	int grid_x = (valid_points_num - 1) / block_x + 1;
	dim3 grid;

	computePointGradients0<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dj_ang_a_, dj_ang_b_, dj_ang_c_, dj_ang_d_,
												point_gradients);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computePointGradients1<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dj_ang_e_, dj_ang_f_, dj_ang_g_, dj_ang_h_,
												point_gradients);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computePointHessian0<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_a2_, dh_ang_a3_,
												dh_ang_b2_, dh_ang_b3_,
												point_hessians);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computePointHessian1<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_c2_, dh_ang_c3_,
												dh_ang_d1_, dh_ang_d2_, dh_ang_d3_,
												point_hessians);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computePointHessian2<<<grid_x, block_x>>>(x_, y_, z_, points_number_,
												valid_points, valid_points_num,
												dh_ang_e1_, dh_ang_e2_, dh_ang_e3_,
												dh_ang_f1_, dh_ang_f2_, dh_ang_f3_,
												point_hessians);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	double *tmp_hessian;

	checkCudaErrors(hipMalloc(&tmp_hessian, sizeof(double) * valid_voxel_num * 6));

	double *e_x_cov_x;

	checkCudaErrors(hipMalloc(&e_x_cov_x, sizeof(double) * valid_voxel_num));

	double *cov_dxd_pi;

	checkCudaErrors(hipMalloc(&cov_dxd_pi, sizeof(double) * valid_voxel_num * 3 * 6));

	computeExCovX<<<grid_x, block_x>>>(trans_x, trans_y, trans_z, valid_points,
										starting_voxel_id, voxel_id, valid_points_num,
										centroid, inverse_covariance, voxel_num,
										gauss_d1_, gauss_d2_,
										e_x_cov_x);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	grid.x = grid_x;
	grid.y = 3;
	grid.z = 6;
	computeCovDxdPi<<<grid, block_x>>>(valid_points, starting_voxel_id, voxel_id, valid_points_num,
											inverse_covariance, voxel_num,
											gauss_d1_, gauss_d2_, point_gradients,
											cov_dxd_pi, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	int block_x2 = (valid_voxel_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : valid_voxel_num;
	int grid_x2 = (valid_voxel_num - 1) / block_x2 + 1;

	updateExCovX<<<grid_x2, block_x2>>>(e_x_cov_x, gauss_d2_, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	grid.y = 6;
	grid.z = 1;
	computeHessianListS0<<<grid_x, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												centroid, inverse_covariance, voxel_num,
												gauss_d1_, gauss_d2_,
												point_gradients, tmp_hessian, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computeHessianListS1<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												centroid, inverse_covariance, voxel_num,
												gauss_d1_, gauss_d2_,
												point_gradients, tmp_hessian, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computeHessianListS2<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												centroid, inverse_covariance, voxel_num,
												gauss_d1_, gauss_d2_,
												point_gradients, tmp_hessian, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	grid.z = 6;
	computeHessianListS3<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												centroid, voxel_num,
												gauss_d1_, gauss_d2_,
												hessians,
												e_x_cov_x, tmp_hessian, cov_dxd_pi,
												valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());


	computeHessianListS4<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												centroid, inverse_covariance, voxel_num,
												gauss_d1_, gauss_d2_,
												point_hessians, hessians,
												e_x_cov_x);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computeHessianListS5<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_hessians, hessians,
													e_x_cov_x);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computeHessianListS6<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
													starting_voxel_id, voxel_id, valid_points_num,
													centroid, inverse_covariance, voxel_num,
													gauss_d1_, gauss_d2_,
													point_hessians, hessians,
													e_x_cov_x);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	computeHessianListS7<<<grid, block_x>>>(trans_x, trans_y, trans_z, valid_points,
												starting_voxel_id, voxel_id, valid_points_num,
												gauss_d1_, gauss_d2_,
												point_gradients, hessians,
												e_x_cov_x, cov_dxd_pi, valid_voxel_num);
	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());

	int full_size = valid_points_num;
	int half_size = (full_size - 1) / 2 + 1;

	while (full_size > 1) {
		block_x = (half_size > BLOCK_SIZE_X) ? BLOCK_SIZE_X : half_size;
		grid_x = (half_size - 1) / block_x + 1;

		grid.x = grid_x;
		grid.y = 6;
		grid.z = 6;
		matrixSum<<<grid_x, block_x>>>(hessians, full_size, half_size, 6, 6, valid_points_num);
//		checkCudaErrors(hipDeviceSynchronize());

		full_size = half_size;
		half_size = (full_size - 1) / 2 + 1;
	}

	checkCudaErrors(hipDeviceSynchronize());

	MatrixDevice dhessian(6, 6, valid_points_num, hessians);
	MatrixHost hhessian(6, 6);

	hhessian.moveToHost(dhessian);

	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 6; j++) {
			hessian(i, j) = hhessian(i, j);
		}
	}

	checkCudaErrors(hipFree(hessians));
	checkCudaErrors(hipFree(point_hessians));
	checkCudaErrors(hipFree(point_gradients));

	checkCudaErrors(hipFree(tmp_hessian));
	checkCudaErrors(hipFree(e_x_cov_x));
	checkCudaErrors(hipFree(cov_dxd_pi));

	if (valid_points != NULL) {
		checkCudaErrors(hipFree(valid_points));
	}

	if (voxel_id != NULL) {
		checkCudaErrors(hipFree(voxel_id));
	}

	if (starting_voxel_id != NULL) {
		checkCudaErrors(hipFree(starting_voxel_id));
	}

	dhessian.memFree();
}


double GNormalDistributionsTransform::getFitnessScore(double max_range)
{
	double fitness_score = 0.0;

	float *trans_x, *trans_y, *trans_z;

	checkCudaErrors(hipMalloc(&trans_x, sizeof(float) * points_number_));
	checkCudaErrors(hipMalloc(&trans_y, sizeof(float) * points_number_));
	checkCudaErrors(hipMalloc(&trans_z, sizeof(float) * points_number_));

	transformPointCloud(x_, y_, z_, trans_x, trans_y, trans_z, points_number_, final_transformation_);

	int *valid_distance;

	checkCudaErrors(hipMalloc(&valid_distance, sizeof(int) * points_number_));

	double *min_distance;

	checkCudaErrors(hipMalloc(&min_distance, sizeof(double) * points_number_));

	voxel_grid_.nearestNeighborSearch(trans_x, trans_y, trans_z, points_number_, valid_distance, min_distance, max_range);

	int size = points_number_;
	int half_size;

	while (size > 1) {
		half_size = (size - 1) / 2 + 1;

		int block_x = (half_size > BLOCK_SIZE_X) ? BLOCK_SIZE_X : half_size;
		int grid_x = (half_size - 1) / block_x + 1;

		gpuSum<double><<<grid_x, block_x>>>(min_distance, size, half_size);
		checkCudaErrors(hipGetLastError());

		gpuSum<int><<<grid_x, block_x>>>(valid_distance, size, half_size);
		checkCudaErrors(hipGetLastError());

		size = half_size;
	}

	checkCudaErrors(hipDeviceSynchronize());

	int nr;

	checkCudaErrors(hipMemcpy(&nr, valid_distance, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&fitness_score, min_distance, sizeof(double), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(trans_x));
	checkCudaErrors(hipFree(trans_y));
	checkCudaErrors(hipFree(trans_z));
	checkCudaErrors(hipFree(valid_distance));
	checkCudaErrors(hipFree(min_distance));

	if (nr > 0)
		return (fitness_score / nr);

	return DBL_MAX;
}

}
