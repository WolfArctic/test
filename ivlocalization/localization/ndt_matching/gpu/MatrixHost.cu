#include "hip/hip_runtime.h"
#include "MatrixHost.h"
#include "debug.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

namespace gpu {

extern "C" __global__ void copyMatrixDevToDev(MatrixDevice input, MatrixDevice output) {
	int row = threadIdx.x;
	int col = threadIdx.y;
	int rows_num = input.rows();
	int cols_num = input.cols();

	if (row < rows_num && col < cols_num)
		output(row, col) = input(row, col);
}

bool MatrixHost::moveToHost(MatrixDevice input){
	if (rows_ != input.rows() || cols_ != input.cols())
		return false;

	if (offset_ == input.offset()) {
		checkCudaErrors(hipMemcpy(buffer_, input.buffer(), sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		return true;
	}
	else {
		double *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		copyMatrixDevToDev << <grid_x, block_x >> >(input, tmp_output);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(buffer_, tmp, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(tmp));

		return true;
	}
}

bool MatrixHost::moveToGpu(MatrixDevice output){
	if (rows_ != output.rows() || cols_ != output.cols())
		return false;

	if (offset_ == output.offset()) {
		checkCudaErrors(hipMemcpy(output.buffer(), buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));
		return true;
	}
	else {
		double *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));
		checkCudaErrors(hipMemcpy(tmp, buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		copyMatrixDevToDev<<<grid_x, block_x>>>(tmp_output, output);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(tmp));

		return true;
	}
}
}
